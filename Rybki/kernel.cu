#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#include <cmath>

__constant__ const float SPEED_FACTOR = 0.1f;
__constant__ const float MARGIN = 0.1f;

// CUDA Kernel
__global__ void fishKernel(const FishArray *in_array, FishArray *out_array, int number_of_fish, float visible_range, float protected_range, float avoid_factor, float matching_factor, float centering_factor, float turn_factor, float min_speed, float max_speed)
{
    int fish_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (fish_idx >= number_of_fish) return;

    Fish fish = {
        in_array->x[fish_idx],
        in_array->y[fish_idx],
        in_array->z[fish_idx],
        in_array->vx[fish_idx],
        in_array->vy[fish_idx],
        in_array->vz[fish_idx]
    };

    Fish otherFish;

    float dx, dy, dz;
    float distance_squared;

    float xpos_avg = 0, ypos_avg = 0, zpos_avg = 0;
    float xvel_avg = 0, yvel_avg = 0, zvel_avg = 0;
    float neighbors = 0;
    float close_dx = 0, close_dy = 0, close_dz = 0;
    float speed;

    for (int i = 0; i < number_of_fish; i++)
    {
        if (i == fish_idx) continue;

        otherFish.x = in_array->x[i];
        otherFish.y = in_array->y[i];
        otherFish.z = in_array->z[i];
        otherFish.vx = in_array->vx[i];
        otherFish.vy = in_array->vy[i];
        otherFish.vz = in_array->vz[i];

        dx = fish.x - otherFish.x;
        dy = fish.y - otherFish.y;
        dz = fish.z - otherFish.z;
        distance_squared = dx * dx + dy * dy + dz * dz;

        // Separation - steer away from another fish in protected range 
        if (distance_squared < protected_range * protected_range)
        {
            close_dx += dx;
            close_dy += dy;
            close_dz += dz;
        }

		// Alignment and Cohesion - steer towards the average position and velocity of other fish in visible range
        if (distance_squared < visible_range * visible_range)
        {
            xpos_avg += otherFish.x;
            ypos_avg += otherFish.y;
            zpos_avg += otherFish.z;
            xvel_avg += otherFish.vx;
            yvel_avg += otherFish.vy;
            zvel_avg += otherFish.vz;
            neighbors++;
        }
    }

	// Alignment and Cohesion - update 
    if (neighbors > 0)
    {
        xpos_avg /= neighbors;
        ypos_avg /= neighbors;
        zpos_avg /= neighbors;
        xvel_avg /= neighbors;
        yvel_avg /= neighbors;
        zvel_avg /= neighbors;

        fish.vx += (xpos_avg - fish.x) * centering_factor + (xvel_avg - fish.vx) * matching_factor;
        fish.vy += (ypos_avg - fish.y) * centering_factor + (yvel_avg - fish.vy) * matching_factor;
        fish.vz += (zpos_avg - fish.z) * centering_factor + (zvel_avg - fish.vz) * matching_factor;
    }

    fish.vx += close_dx * avoid_factor;
    fish.vy += close_dy * avoid_factor;
    fish.vz += close_dz * avoid_factor;

	// turn the fish if it is too close to the edge
    if (fish.x < MARGIN - 1)
    {
        fish.vx += turn_factor;
    }
    else if (fish.x > 1 - MARGIN)
    {
        fish.vx -= turn_factor;
    }
    if (fish.y < MARGIN - 1)
    {
        fish.vy += turn_factor;
    }
    else if (fish.y > 1 - MARGIN)
    {
        fish.vy -= turn_factor;
    }
    if (fish.z < MARGIN - 1)
    {
        fish.vz += turn_factor;
    }
    else if (fish.z > 1 - MARGIN)
    {
        fish.vz -= turn_factor;
    }

	// limit the speed of the fish
    speed = sqrtf(fish.vx * fish.vx + fish.vy * fish.vy + fish.vz * fish.vz);
    if (speed < min_speed)
    {
        fish.vx = (fish.vx / speed) * min_speed;
        fish.vy = (fish.vy / speed) * min_speed;
        fish.vz = (fish.vz / speed) * min_speed;
    }
    else if (speed > max_speed)
    {
        fish.vx = (fish.vx / speed) * max_speed;
        fish.vy = (fish.vy / speed) * max_speed;
        fish.vz = (fish.vz / speed) * max_speed;
    }

    fish.x += fish.vx * SPEED_FACTOR;
    fish.y += fish.vy * SPEED_FACTOR;
    fish.z += fish.vz * SPEED_FACTOR;

    out_array->x[fish_idx] = fish.x;
    out_array->y[fish_idx] = fish.y;
    out_array->z[fish_idx] = fish.z;
    out_array->vx[fish_idx] = fish.vx;
    out_array->vy[fish_idx] = fish.vy;
    out_array->vz[fish_idx] = fish.vz;
}

// Helper function for using CUDA to update fish positions in parallel.
hipError_t update_fish_positions_cuda(FishArray *arr, FishArray *dev_old, FishArray *dev_new, int number_of_fish, float visible_range, float protected_range, float avoid_factor, float matching_factor, float centering_factor, float turn_factor, float min_speed, float max_speed)
{

    hipError_t cudaStatus;

	// calculate the number of blocks and threads
    int THREADS_PER_BLOCK = number_of_fish > 512 ? 512 : number_of_fish;
	int NUMBER_OF_BLOCKS = (number_of_fish + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch the kernel with updated parameters.
    fishKernel << <NUMBER_OF_BLOCKS, THREADS_PER_BLOCK >> > (dev_old, dev_new, number_of_fish, visible_range, protected_range, avoid_factor, matching_factor, centering_factor, turn_factor, min_speed, max_speed);

    // Check for any errors launching the kernel.
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "fishKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching fishKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output array from GPU buffer to host memory.
	cudaStatus = copy_fish_array_device_to_host(arr, dev_new, number_of_fish);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    std::swap(dev_old, dev_new);

    return cudaStatus;

Error:

    free_fish_array(dev_old);
	free_fish_array(dev_new);

    return cudaStatus;
}

hipError_t allocate_fish_array(FishArray *arr, int number_of_fish)
{
    hipError_t cudaStatus;

    // Allocate GPU buffers for input and output arrays.
	cudaStatus = hipMalloc((void **)&arr->x, number_of_fish * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void **)&arr->y, number_of_fish * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&arr->z, number_of_fish * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&arr->vx, number_of_fish * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&arr->vy, number_of_fish * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&arr->vz, number_of_fish * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    return cudaStatus;

Error:
	hipFree(arr->x);
	hipFree(arr->y);
	hipFree(arr->z);
	hipFree(arr->vx);
	hipFree(arr->vy);
	hipFree(arr->vz);
    
	return cudaStatus;
}


hipError_t free_fish_array(FishArray *arr)
{
    hipError_t cudaStatus;
    cudaStatus = hipFree(arr->x);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipFree failed!");
        goto Error;
    }
    cudaStatus = hipFree(arr->y);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipFree failed!");
        goto Error;
    }
    cudaStatus = hipFree(arr->z);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipFree failed!");
        goto Error;
    }
    cudaStatus = hipFree(arr->vx);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipFree failed!");
        goto Error;
    }
    cudaStatus = hipFree(arr->vy);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipFree failed!");
        goto Error;
    }
    cudaStatus = hipFree(arr->vz);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipFree failed!");
        goto Error;
    }

    return cudaStatus;

Error:
    hipFree(arr->x);
    hipFree(arr->y);
    hipFree(arr->z);
    hipFree(arr->vx);
    hipFree(arr->vy);
    hipFree(arr->vz);

    return cudaStatus;
}

hipError_t initial_copy(FishArray *dev_arr, FishArray arr, int number_of_fish)
{
	hipError_t cudaStatus;

	cudaStatus = hipMemcpy(dev_arr->x, arr.x, number_of_fish * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_arr->y, arr.y, number_of_fish * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_arr->z, arr.z, number_of_fish * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_arr->vx, arr.vx, number_of_fish * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_arr->vy, arr.vy, number_of_fish * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_arr->vz, arr.vz, number_of_fish * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    return cudaStatus;

Error:
	free_fish_array(dev_arr);

    return cudaStatus;
}


hipError_t copy_fish_array_device_to_host(FishArray *arr, FishArray *dev_arr, int number_of_fish)
{
    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(arr->x, dev_arr->x, number_of_fish * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(arr->y, dev_arr->y, number_of_fish * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(arr->z, dev_arr->z, number_of_fish * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(arr->vx, dev_arr->vx, number_of_fish * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(arr->vy, dev_arr->vy, number_of_fish * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(arr->vz, dev_arr->vz, number_of_fish * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    return cudaStatus;

Error:
    free_fish_array(dev_arr);
	return cudaStatus;
}